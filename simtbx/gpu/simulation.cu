#include "hip/hip_runtime.h"
#include <scitbx/array_family/boost_python/flex_fwd.h>
#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/simulation.h>
#include <simtbx/gpu/simulation.cuh>
#include <scitbx/array_family/flex_types.h>
#define THREADS_PER_BLOCK_X 128
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_TOTAL (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)

namespace simtbx {
namespace gpu {

namespace af = scitbx::af;
//refactor later into helper file
  static hipError_t cudaMemcpyVectorDoubleToDevice(CUDAREAL *dst, const double *src, size_t vector_items) {
	CUDAREAL * temp = new CUDAREAL[vector_items];
	for (size_t i = 0; i < vector_items; i++) {
		temp[i] = src[i];
	}
	hipError_t ret = hipMemcpy(dst, temp, sizeof(*dst) * vector_items, hipMemcpyHostToDevice);
	delete temp;
	return ret;
  }

/* make a unit vector pointing in same direction and report magnitude (both args can be same vector) */
  double cpu_unitize(const double * vector, double * new_unit_vector) {

	double v1 = vector[1];
	double v2 = vector[2];
	double v3 = vector[3];

	double mag = sqrt(v1 * v1 + v2 * v2 + v3 * v3);

	if (mag != 0.0) {
		/* normalize it */
		new_unit_vector[0] = mag;
		new_unit_vector[1] = v1 / mag;
		new_unit_vector[2] = v2 / mag;
		new_unit_vector[3] = v3 / mag;
	} else {
		/* can't normalize, report zero vector */
		new_unit_vector[0] = 0.0;
		new_unit_vector[1] = 0.0;
		new_unit_vector[2] = 0.0;
		new_unit_vector[3] = 0.0;
	}
	return mag;
  }

  void
  exascale_api::show(){
    std::cout << "SIM.phisteps" << SIM.phisteps << std::endl;
  }

  void 
  exascale_api::add_energy_channel_from_gpu_amplitudes_cuda(
    int const& ichannel,
    simtbx::gpu::gpu_energy_channels & gec,
    simtbx::gpu::gpu_detector & gdt
  ){
        cudaSafeCall(hipSetDevice(SIM.device_Id));

        // transfer source_I, source_lambda
        // the int arguments are for sizes of the arrays
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, SIM.sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, SIM.sources));

        // magic happens here: take pointer from singleton, temporarily use it for add Bragg iteration:
        cu_current_channel_Fhkl = gec.d_channel_Fhkl[ichannel];

        hipDeviceProp_t deviceProps = { 0 };
        cudaSafeCall(hipGetDeviceProperties(&deviceProps, SIM.device_Id));
        int smCount = deviceProps.multiProcessorCount;
        dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
        dim3 numBlocks(smCount * 8, 1);

        // want to loop thru panels and increment the array ptrs XXX FIXME
        nanoBraggSpotsCUDAKernel<<<numBlocks, threadsPerBlock>>>(
          gdt.cu_slow_pixels, gdt.cu_fast_pixels, SIM.roi_xmin,
          SIM.roi_xmax, SIM.roi_ymin, SIM.roi_ymax, SIM.oversample, SIM.point_pixel,
          SIM.pixel_size, cu_subpixel_size, cu_steps, SIM.detector_thickstep, SIM.detector_thicksteps,
          SIM.detector_thick, SIM.detector_attnlen, cu_sdet_vector, cu_fdet_vector, cu_odet_vector,
          cu_pix0_vector, SIM.curved_detector, SIM.distance, SIM.close_distance, cu_beam_vector,
          SIM.Xbeam, SIM.Ybeam, SIM.dmin, SIM.phi0, SIM.phistep, SIM.phisteps, cu_spindle_vector,
          SIM.sources, cu_source_X, cu_source_Y, cu_source_Z,
          cu_source_I, cu_source_lambda, cu_a0, cu_b0,
          cu_c0, SIM.xtal_shape, SIM.mosaic_spread, SIM.mosaic_domains, cu_mosaic_umats,
          SIM.Na, SIM.Nb, SIM.Nc, SIM.V_cell,
          cu_water_size, cu_water_F, cu_water_MW, simtbx::nanoBragg::r_e_sqr, SIM.fluence,
          simtbx::nanoBragg::Avogadro, SIM.spot_scale, SIM.integral_form, SIM.default_F,
          SIM.interpolate, cu_current_channel_Fhkl, gec.cu_FhklParams, SIM.nopolar,
          cu_polar_vector, SIM.polarization, SIM.fudge,
          gdt.cu_maskimage, gdt.cu_floatimage /*out*/, gdt.cu_omega_reduction/*out*/,
          gdt.cu_max_I_x_reduction/*out*/, gdt.cu_max_I_y_reduction /*out*/, gdt.cu_rangemap /*out*/);

        //don't want to free the gec data when the nanoBragg goes out of scope, so switch the pointer
        cu_current_channel_Fhkl = NULL;

        cudaSafeCall(hipPeekAtLastError());
        cudaSafeCall(hipDeviceSynchronize());

        add_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(gdt.cu_accumulate_floatimage, gdt.cu_floatimage,
          gdt.cu_n_panels * gdt.cu_slow_pixels * gdt.cu_fast_pixels);
  }

  void
  exascale_api::add_background_cuda(simtbx::gpu::gpu_detector & gdt){
        cudaSafeCall(hipSetDevice(SIM.device_Id));

        // transfer source_I, source_lambda
        // the int arguments are for sizes of the arrays
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, SIM.sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, SIM.sources));

        CUDAREAL * cu_stol_of;
        cudaSafeCall(hipMalloc((void ** )&cu_stol_of, sizeof(*cu_stol_of) * SIM.stols));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_stol_of, SIM.stol_of, SIM.stols));

        CUDAREAL * cu_Fbg_of;
        cudaSafeCall(hipMalloc((void ** )&cu_Fbg_of, sizeof(*cu_Fbg_of) * SIM.stols));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_Fbg_of, SIM.Fbg_of, SIM.stols));

        hipDeviceProp_t deviceProps = { 0 };
        cudaSafeCall(hipGetDeviceProperties(&deviceProps, SIM.device_Id));
        int smCount = deviceProps.multiProcessorCount;
        dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
        dim3 numBlocks(smCount * 8, 1);

        // the for loop around panels will go here.  Offsets will be given.

        //  initialize the device memory within a kernel. //havn't analyzed to see if initialization is needed
        nanoBraggSpotsInitCUDAKernel<<<numBlocks, threadsPerBlock>>>(
          gdt.cu_slow_pixels, gdt.cu_fast_pixels,
          gdt.cu_floatimage, gdt.cu_omega_reduction, gdt.cu_max_I_x_reduction, gdt.cu_max_I_y_reduction,
          gdt.cu_rangemap);
        cudaSafeCall(hipPeekAtLastError());
        cudaSafeCall(hipDeviceSynchronize());

        add_background_CUDAKernel<<<numBlocks, threadsPerBlock>>>(SIM.sources, SIM.oversample,
          SIM.pixel_size, gdt.cu_slow_pixels, gdt.cu_fast_pixels, SIM.detector_thicksteps,
          SIM.detector_thickstep, SIM.detector_attnlen,
          cu_sdet_vector, cu_fdet_vector, cu_odet_vector, cu_pix0_vector,
          SIM.close_distance, SIM.point_pixel, SIM.detector_thick,
          cu_source_X, cu_source_Y, cu_source_Z,
          cu_source_lambda, cu_source_I,
          SIM.stols, cu_stol_of, cu_Fbg_of,
          SIM.nopolar, SIM.polarization, cu_polar_vector,
          simtbx::nanoBragg::r_e_sqr, SIM.fluence, SIM.amorphous_molecules,
          gdt.cu_floatimage /*out*/);

        cudaSafeCall(hipPeekAtLastError());
        cudaSafeCall(hipDeviceSynchronize());
        add_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(gdt.cu_accumulate_floatimage, gdt.cu_floatimage,
          gdt.cu_n_panels * gdt.cu_slow_pixels * gdt.cu_fast_pixels);

        cudaSafeCall(hipFree(cu_stol_of));
        cudaSafeCall(hipFree(cu_Fbg_of));
}

  void
  exascale_api::allocate_cuda(){
    cudaSafeCall(hipSetDevice(SIM.device_Id));

    /* water_size not defined in class, CLI argument, defaults to 0 */
    double water_size = 0.0;
    /* missing constants */
    double water_F = 2.57;
    double water_MW = 18.0;

    /* make sure we are normalizing with the right number of sub-steps */
    int nb_steps = SIM.phisteps*SIM.mosaic_domains*SIM.oversample*SIM.oversample;
    double nb_subpixel_size = SIM.pixel_size/SIM.oversample;

        /*create transfer arguments to device space*/
        cu_subpixel_size = nb_subpixel_size; //check for conflict?
        cu_steps = nb_steps; //check for conflict?

        /* presumably thickness and attenuation can be migrated to the gpu detector class XXX FIXME*/
        //cu_detector_thick = SIM.detector_thick;
        //cu_detector_mu = SIM.detector_attnlen; // synonyms
        //cu_distance = SIM.distance; /* distance and close distance, detector properties? XXX FIXME */
        //cu_close_distance = SIM.close_distance;

        cu_water_size = water_size;
        cu_water_F = water_F;
        cu_water_MW = water_MW;

        const int vector_length = 4;
        int cu_sources = SIM.sources;
        int cu_mosaic_domains = SIM.mosaic_domains;

        /* presumably should come from detector class */
        cudaSafeCall(hipMalloc((void ** )&cu_sdet_vector, sizeof(*cu_sdet_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_sdet_vector, SIM.sdet_vector, vector_length));

        /* presumably should come from detector class */
        cudaSafeCall(hipMalloc((void ** )&cu_fdet_vector, sizeof(*cu_fdet_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_fdet_vector, SIM.fdet_vector, vector_length));

        /* presumably should come from detector class */
        cudaSafeCall(hipMalloc((void ** )&cu_odet_vector, sizeof(*cu_odet_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_odet_vector, SIM.odet_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_pix0_vector, sizeof(*cu_pix0_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_pix0_vector, SIM.pix0_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_beam_vector, sizeof(*cu_beam_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_beam_vector, SIM.beam_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_spindle_vector, sizeof(*cu_spindle_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_spindle_vector, SIM.spindle_vector, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_a0, sizeof(*cu_a0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_a0, SIM.a0, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_b0, sizeof(*cu_b0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_b0, SIM.b0, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_c0, sizeof(*cu_c0) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_c0, SIM.c0, vector_length));

        // Unitize polar vector before sending it to the GPU. 
        // Optimization do it only once here rather than multiple time per pixel in the GPU.
        double polar_vector_unitized[4];
        cpu_unitize(SIM.polar_vector, polar_vector_unitized);
        cudaSafeCall(hipMalloc((void ** )&cu_polar_vector, sizeof(*cu_polar_vector) * vector_length));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_polar_vector, polar_vector_unitized, vector_length));

        cudaSafeCall(hipMalloc((void ** )&cu_source_X, sizeof(*cu_source_X) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_X, SIM.source_X, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_Y, sizeof(*cu_source_Y) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_Y, SIM.source_Y, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_Z, sizeof(*cu_source_Z) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_Z, SIM.source_Z, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_I, sizeof(*cu_source_I) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_I, SIM.source_I, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_source_lambda, sizeof(*cu_source_lambda) * cu_sources));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, SIM.source_lambda, cu_sources));

        cudaSafeCall(hipMalloc((void ** )&cu_mosaic_umats, sizeof(*cu_mosaic_umats) * cu_mosaic_domains * 9));
        cudaSafeCall(cudaMemcpyVectorDoubleToDevice(cu_mosaic_umats, SIM.mosaic_umats, cu_mosaic_domains * 9));
  };

  exascale_api::~exascale_api(){
    cudaSafeCall(hipSetDevice(SIM.device_Id));

        cudaSafeCall(hipFree(cu_sdet_vector));
        cudaSafeCall(hipFree(cu_fdet_vector));
        cudaSafeCall(hipFree(cu_odet_vector));
        cudaSafeCall(hipFree(cu_pix0_vector));
        cudaSafeCall(hipFree(cu_beam_vector));
        cudaSafeCall(hipFree(cu_spindle_vector));
        cudaSafeCall(hipFree(cu_source_X));
        cudaSafeCall(hipFree(cu_source_Y));
        cudaSafeCall(hipFree(cu_source_Z));
        cudaSafeCall(hipFree(cu_source_I));
        cudaSafeCall(hipFree(cu_source_lambda));
        cudaSafeCall(hipFree(cu_a0));
        cudaSafeCall(hipFree(cu_b0));
        cudaSafeCall(hipFree(cu_c0));
        cudaSafeCall(hipFree(cu_mosaic_umats));
        cudaSafeCall(hipFree(cu_polar_vector));
  }

} // gpu
} // simtbx
