#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/detector.h>

namespace simtbx {
namespace gpu {

  gpu_detector::gpu_detector(int const& arg_device_id, 
                             dxtbx::model::Detector const & arg_detector):
    h_deviceID(arg_device_id),
    detector(arg_detector),
    cu_accumulate_floatimage(NULL) {
    hipSetDevice(arg_device_id);

    //1) determine the size
    cu_n_panels = detector.size();
    SCITBX_ASSERT( cu_n_panels >= 1);

    //2) confirm that array dimensions are similar for each size
    cu_slow_pixels = detector[0].get_image_size()[0];
    cu_fast_pixels = detector[0].get_image_size()[1];
    for (int ipanel=1; ipanel < detector.size(); ++ipanel){
      SCITBX_ASSERT(detector[ipanel].get_image_size()[0] == cu_slow_pixels);
      SCITBX_ASSERT(detector[ipanel].get_image_size()[1] == cu_fast_pixels);
    }
    _image_size = cu_n_panels * cu_slow_pixels * cu_fast_pixels;

    //3) allocate a cuda array with these dimensions
    /* separate accumulator image outside the usual nanoBragg data structure.
           1. accumulate contributions from a sequence of source energy channels computed separately
           2. represent multiple panels, all same rectangular shape; slowest dimension = n_panels */
    cudaSafeCall(hipMalloc((void ** )&cu_accumulate_floatimage, 
                            sizeof(*cu_accumulate_floatimage) * _image_size));
    cudaSafeCall(hipMemset((void *)cu_accumulate_floatimage, 0, 
                            sizeof(*cu_accumulate_floatimage) * _image_size));
    std::cout<<"allocated cuda array of "<<_image_size<<std::endl;
  };

  void gpu_detector::free_detail(){
    hipSetDevice(h_deviceID);
    //4) make sure we can deallocate cuda array later on
    if (cu_accumulate_floatimage != NULL) {
      cudaSafeCall(hipFree(cu_accumulate_floatimage));
    }
  };

  void
  gpu_detector::each_image_allocate_cuda(){
    hipSetDevice(h_deviceID);
    /*allocate and zero reductions */
    bool * rangemap = (bool*) calloc(_image_size, sizeof(bool));
    float * omega_reduction = (float*) calloc(_image_size, sizeof(float));
    float * max_I_x_reduction = (float*) calloc(_image_size, sizeof(float));
    float * max_I_y_reduction = (float*) calloc(_image_size, sizeof(float));
    //It is not quite clear why we must zero them on CPU, why not just on GPU?
    
    cu_omega_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_omega_reduction, sizeof(*cu_omega_reduction) * _image_size));
    printf("Allocated %d bytes to \n",sizeof(*cu_omega_reduction) * _image_size);
    std::cout<< cu_omega_reduction<<std::endl;
    cudaSafeCall(hipMemcpy(cu_omega_reduction, 
                 omega_reduction, sizeof(*cu_omega_reduction) * _image_size, 
                 hipMemcpyHostToDevice));

    cu_max_I_x_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * _image_size));
    cudaSafeCall(hipMemcpy(cu_max_I_x_reduction, 
                 max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * _image_size,
                 hipMemcpyHostToDevice));

    cu_max_I_y_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * _image_size));
    cudaSafeCall(hipMemcpy(cu_max_I_y_reduction, max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * _image_size,
                 hipMemcpyHostToDevice));

    cu_rangemap = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_rangemap, sizeof(*cu_rangemap) * _image_size));
    cudaSafeCall(hipMemcpy(cu_rangemap, 
                 rangemap, sizeof(*cu_rangemap) * _image_size,
                 hipMemcpyHostToDevice));

    // deallocate host arrays
    // potential memory leaks
    free(rangemap);
    free(omega_reduction);
    free(max_I_x_reduction);
    free(max_I_y_reduction);

    cu_maskimage = NULL;
    int unsigned short * maskimage = NULL; //default case, must implement non-trivial initializer elsewhere
    if (maskimage != NULL) {
      cudaSafeCall(hipMalloc((void ** )&cu_maskimage, sizeof(*cu_maskimage) * _image_size));
      cudaSafeCall(hipMemcpy(cu_maskimage, maskimage, sizeof(*cu_maskimage) * _image_size,
                   hipMemcpyHostToDevice));
    }

    // In contrast to old API, new API initializes its own accumulator, does not take values from CPU
    cu_floatimage = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_floatimage, sizeof(*cu_floatimage) * _image_size));

  }

  void
  gpu_detector::each_image_free_cuda(){
    hipSetDevice(h_deviceID);
    SCITBX_EXAMINE(h_deviceID);
    SCITBX_EXAMINE(cu_omega_reduction);
    SCITBX_EXAMINE(cu_max_I_x_reduction);
    SCITBX_EXAMINE(cu_max_I_y_reduction);
    SCITBX_EXAMINE(cu_rangemap);
    SCITBX_EXAMINE(cu_maskimage);
    SCITBX_EXAMINE(cu_floatimage);
    cudaSafeCall(hipDeviceSynchronize());
Line:  112
Error: misaligned address
    cudaSafeCall(hipFree(cu_omega_reduction));
    cudaSafeCall(hipFree(cu_max_I_x_reduction));
    cudaSafeCall(hipFree(cu_max_I_y_reduction));
    cudaSafeCall(hipFree(cu_rangemap));
    cudaSafeCall(hipFree(cu_maskimage));
    cudaSafeCall(hipFree(cu_floatimage));
  }

} // gpu
} // simtbx
