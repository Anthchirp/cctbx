/*#include <scitbx/array_family/boost_python/flex_fwd.h>*/
#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/detector.h>
#include <simtbx/gpu/detector.cuh>
#define THREADS_PER_BLOCK_X 128
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_TOTAL (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)

namespace simtbx {
namespace gpu {

  gpu_detector::gpu_detector(int const& arg_device_id,
                             dxtbx::model::Detector const & arg_detector):
    h_deviceID(arg_device_id),
    detector(arg_detector),
    cu_accumulate_floatimage(NULL) {
    hipSetDevice(arg_device_id);

    //1) determine the size
    cu_n_panels = detector.size();
    SCITBX_ASSERT( cu_n_panels >= 1);

    //2) confirm that array dimensions are similar for each size
    cu_slow_pixels = detector[0].get_image_size()[0];
    cu_fast_pixels = detector[0].get_image_size()[1];
    for (int ipanel=1; ipanel < detector.size(); ++ipanel){
      SCITBX_ASSERT(detector[ipanel].get_image_size()[0] == cu_slow_pixels);
      SCITBX_ASSERT(detector[ipanel].get_image_size()[1] == cu_fast_pixels);
    }
    _image_size = cu_n_panels * cu_slow_pixels * cu_fast_pixels;

    //3) allocate a cuda array with these dimensions
    /* separate accumulator image outside the usual nanoBragg data structure.
           1. accumulate contributions from a sequence of source energy channels computed separately
           2. represent multiple panels, all same rectangular shape; slowest dimension = n_panels */
    cudaSafeCall(hipMalloc((void ** )&cu_accumulate_floatimage,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
    cudaSafeCall(hipMemset((void *)cu_accumulate_floatimage, 0,
                            sizeof(*cu_accumulate_floatimage) * _image_size));
  };

  void gpu_detector::free_detail(){
    hipSetDevice(h_deviceID);
    //4) make sure we can deallocate cuda array later on
    if (cu_accumulate_floatimage != NULL) {
      cudaSafeCall(hipFree(cu_accumulate_floatimage));
    }
  };

  void
  gpu_detector::scale_in_place_cuda(const double& factor){
    cudaSafeCall(hipSetDevice(h_deviceID));
    hipDeviceProp_t deviceProps = { 0 };
    cudaSafeCall(hipGetDeviceProperties(&deviceProps, h_deviceID));
  int smCount = deviceProps.multiProcessorCount;
  dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
  dim3 numBlocks(smCount * 8, 1);
  int total_pixels = _image_size;
  scale_array_CUDAKernel<<<numBlocks, threadsPerBlock>>>(
    factor, cu_accumulate_floatimage, total_pixels);
  }

  void
  gpu_detector::write_raw_pixels_cuda(simtbx::nanoBragg::nanoBragg& nB){
    //only implement the monolithic detector case, one panel
    SCITBX_ASSERT(nB.spixels == cu_slow_pixels);
    SCITBX_ASSERT(nB.fpixels == cu_fast_pixels);
    SCITBX_ASSERT(cu_n_panels == 1);
    /* nB.raw_pixels = af::flex_double(af::flex_grid<>(nB.spixels,nB.fpixels));
       do not reallocate CPU memory for the data write, as it is not needed
     */
    double * double_floatimage = nB.raw_pixels.begin();
    cudaSafeCall(hipSetDevice(nB.device_Id));
    cudaSafeCall(hipMemcpy(
     double_floatimage,
     cu_accumulate_floatimage,
     sizeof(*cu_accumulate_floatimage) * _image_size,
     hipMemcpyDeviceToHost));
  }

  void
  gpu_detector::each_image_allocate_cuda(){
    hipSetDevice(h_deviceID);
    /*allocate and zero reductions */
    bool * rangemap = (bool*) calloc(_image_size, sizeof(bool));
    float * omega_reduction = (float*) calloc(_image_size, sizeof(float));
    float * max_I_x_reduction = (float*) calloc(_image_size, sizeof(float));
    float * max_I_y_reduction = (float*) calloc(_image_size, sizeof(float));
    //It is not quite clear why we must zero them on CPU, why not just on GPU?

    cu_omega_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_omega_reduction, sizeof(*cu_omega_reduction) * _image_size));
    cudaSafeCall(hipMemcpy(cu_omega_reduction,
                 omega_reduction, sizeof(*cu_omega_reduction) * _image_size,
                 hipMemcpyHostToDevice));

    cu_max_I_x_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * _image_size));
    cudaSafeCall(hipMemcpy(cu_max_I_x_reduction,
                 max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * _image_size,
                 hipMemcpyHostToDevice));

    cu_max_I_y_reduction = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * _image_size));
    cudaSafeCall(hipMemcpy(cu_max_I_y_reduction, max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * _image_size,
                 hipMemcpyHostToDevice));

    cu_rangemap = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_rangemap, sizeof(*cu_rangemap) * _image_size));
    cudaSafeCall(hipMemcpy(cu_rangemap,
                 rangemap, sizeof(*cu_rangemap) * _image_size,
                 hipMemcpyHostToDevice));

    // deallocate host arrays
    // potential memory leaks
    free(rangemap);
    free(omega_reduction);
    free(max_I_x_reduction);
    free(max_I_y_reduction);

    cu_maskimage = NULL;
    int unsigned short * maskimage = NULL; //default case, must implement non-trivial initializer elsewhere
    if (maskimage != NULL) {
      cudaSafeCall(hipMalloc((void ** )&cu_maskimage, sizeof(*cu_maskimage) * _image_size));
      cudaSafeCall(hipMemcpy(cu_maskimage, maskimage, sizeof(*cu_maskimage) * _image_size,
                   hipMemcpyHostToDevice));
    }

    // In contrast to old API, new API initializes its own accumulator, does not take values from CPU
    cu_floatimage = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_floatimage, sizeof(*cu_floatimage) * _image_size));

  }

  void
  gpu_detector::each_image_free_cuda(){
    hipSetDevice(h_deviceID);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipFree(cu_omega_reduction));
    cudaSafeCall(hipFree(cu_max_I_x_reduction));
    cudaSafeCall(hipFree(cu_max_I_y_reduction));
    cudaSafeCall(hipFree(cu_rangemap));
    cudaSafeCall(hipFree(cu_maskimage));
    cudaSafeCall(hipFree(cu_floatimage));
  }

} // gpu
} // simtbx
