#include "hip/hip_runtime.h"
#include "diffBraggCUDA.h"

__global__
void gpu_sum_over_steps(
        int Npix_to_model, unsigned int* panels_fasts_slows,
        CUDAREAL* floatimage,
        CUDAREAL* d_Umat_images, CUDAREAL* d2_Umat_images,
        CUDAREAL* d_Bmat_images, CUDAREAL* d2_Bmat_images,
        CUDAREAL* d_Ncells_images, CUDAREAL* d2_Ncells_images,
        CUDAREAL* d_fcell_images, CUDAREAL* d2_fcell_images,
        CUDAREAL* d_eta_images,
        CUDAREAL* d_lambda_images, CUDAREAL* d2_lambda_images,
        CUDAREAL* d_panel_rot_images, CUDAREAL* d2_panel_rot_images,
        CUDAREAL* d_panel_orig_images, CUDAREAL* d2_panel_orig_images,
        CUDAREAL* d_sausage_XYZ_scale_images,
        const int* __restrict__ subS_pos, const int* __restrict__ subF_pos, const int*  __restrict__ thick_pos,
        const int* __restrict__ source_pos, const int* __restrict__ phi_pos, const int* __restrict__ mos_pos, const int* __restrict__ sausage_pos,
        const int Nsteps, int _printout_fpixel, int _printout_spixel, bool _printout, CUDAREAL _default_F,
        int oversample, bool _oversample_omega, CUDAREAL subpixel_size, CUDAREAL pixel_size,
        CUDAREAL detector_thickstep, CUDAREAL _detector_thick, CUDAREAL close_distance, CUDAREAL detector_attnlen,
        int detector_thicksteps, int sources, int phisteps, int mosaic_domains,
        bool use_lambda_coefficients, CUDAREAL lambda0, CUDAREAL lambda1,
        MAT3 eig_U, MAT3 eig_O, MAT3 eig_B, MAT3 RXYZ,
        VEC3* dF_vecs,
        VEC3* dS_vecs,
        const MAT3* __restrict__ UMATS_RXYZ,
        MAT3* UMATS_RXYZ_prime,
        MAT3* RotMats,
        MAT3* dRotMats,
        MAT3* d2RotMats,
        MAT3* UMATS,
        MAT3* dB_mats,
        MAT3* dB2_mats,
        MAT3* Amatrices,
        MAT3* sausages_RXYZ, MAT3* d_sausages_RXYZ, const MAT3* __restrict__ sausages_U,
        const CUDAREAL* __restrict__ sausages_scale,
        const CUDAREAL* __restrict__ source_X, const CUDAREAL* __restrict__ source_Y,
        const CUDAREAL* __restrict__ source_Z, const CUDAREAL* __restrict__ source_lambda,
        const CUDAREAL* __restrict__ source_I,
        CUDAREAL kahn_factor,
        CUDAREAL Na, CUDAREAL Nb, CUDAREAL Nc,
        CUDAREAL phi0, CUDAREAL phistep,
        VEC3 spindle_vec, VEC3 _polarization_axis,
        int h_range, int k_range, int l_range,
        int h_max, int h_min, int k_max, int k_min, int l_max, int l_min, CUDAREAL dmin,
        CUDAREAL fudge, bool complex_miller, int verbose, bool only_save_omega_kahn,
        bool isotropic_ncells, bool compute_curvatures,
        const CUDAREAL* __restrict__ _FhklLinear, const CUDAREAL* __restrict__ _Fhkl2Linear,
        bool* refine_Bmat, bool* refine_Ncells, bool* refine_panel_origin, bool* refine_panel_rot,
        bool refine_fcell, bool* refine_lambda, bool refine_eta, bool* refine_Umat,
        bool refine_sausages, int num_sausages,
        const CUDAREAL* __restrict__ fdet_vectors, const CUDAREAL* __restrict__ sdet_vectors,
        const CUDAREAL* __restrict__ odet_vectors, const CUDAREAL* __restrict__ pix0_vectors,
        bool _nopolar, bool _point_pixel, CUDAREAL _fluence, CUDAREAL _r_e_sqr, CUDAREAL _spot_scale, int Npanels)
{ // BEGIN GPU kernel

    //extern __shared__ CUDAREAL detector_vectors[];
    //int stride = Npanels*3; // detector vectors stride in shared mem

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //if (tid==0){
    //    for (int i=0; i<stride; i++){
    //        detector_vectors[i] = fdet_vectors[i];
    //        detector_vectors[stride+i] = sdet_vectors[i];
    //        detector_vectors[2*stride+i] = odet_vectors[i];
    //        detector_vectors[3*stride+i] = pix0_vectors[i];
    //    }
    //}
    //__syncthreads();
    int thread_stride = blockDim.x * gridDim.x;
    __shared__ MAT3 s_Ot;
    __shared__ MAT3 s_Amat;
    MAT3 Bmat_realspace = eig_B*1e10;
    if (threadIdx.x==0){
        s_Ot = eig_O.transpose();
        s_Amat = eig_U*eig_B*1e10*s_Ot;
    }
    __syncthreads();

    MAT3 _NABC;
    _NABC << Na,0,0,
            0,Nb,0,
            0,0,Nc;
    CUDAREAL C = 2 / 0.63 * fudge;
    CUDAREAL two_C = 2*C;
    //MAT3 U;
    //U << 1,0,0,
    //     0,1,0,
    //     0,0,1; //= sausages_U[_sausage_tic] * eig_U;
    //__syncthreads();

    for (int i_pix=tid; i_pix < Npix_to_model; i_pix+= thread_stride){
       int _pid = panels_fasts_slows[i_pix*3];
       int _fpixel = panels_fasts_slows[i_pix*3+1];
       int _spixel = panels_fasts_slows[i_pix*3+2];

       // reset photon count for this pixel
       CUDAREAL _I=0;

       // reset derivative photon counts for the various parameters
       CUDAREAL rot_manager_dI[3] = {0,0,0};
       CUDAREAL rot_manager_dI2[3] = {0,0,0};
       CUDAREAL ucell_manager_dI[6]= {0,0,0,0,0,0};
       CUDAREAL ucell_manager_dI2[6]= {0,0,0,0,0,0};
       CUDAREAL Ncells_manager_dI[3]= {0,0,0};
       CUDAREAL Ncells_manager_dI2[3]= {0,0,0};
       CUDAREAL pan_orig_manager_dI[3]= {0,0,0};
       CUDAREAL pan_orig_manager_dI2[3]= {0,0,0};
       CUDAREAL pan_rot_manager_dI[3]= {0,0,0};
       CUDAREAL pan_rot_manager_dI2[3]= {0,0,0};
       CUDAREAL fcell_manager_dI=0;
       CUDAREAL fcell_manager_dI2=0;
       CUDAREAL eta_manager_dI = 0;
       CUDAREAL lambda_manager_dI[2] = {0,0};
       CUDAREAL lambda_manager_dI2[2] = {0,0};

       CUDAREAL sausage_manager_dI[24] = {0,0,0,0,0, // TODO use shared memory determined at runtime to increase max sausages
                                          0,0,0,0,0,
                                          0,0,0,0,0,
                                          0,0,0,0,0,
                                          0,0,0,0}; // maximum of 6 sausages!

       for(int _subS=0;_subS<oversample;++_subS){
       for(int _subF=0;_subF<oversample;++_subF){

           // absolute mm position on detector (relative to its origin)
           CUDAREAL _Fdet = subpixel_size*(_fpixel*oversample + _subF ) + subpixel_size/2.0;
           CUDAREAL _Sdet = subpixel_size*(_spixel*oversample + _subS ) + subpixel_size/2.0;

           // assume "distance" is to the front of the detector sensor layer
           int pid_x = _pid*3;
           int pid_y = _pid*3+1;
           int pid_z = _pid*3+2;

           CUDAREAL fx = fdet_vectors[pid_x];
           CUDAREAL fy = fdet_vectors[pid_y];
           CUDAREAL fz = fdet_vectors[pid_z];

           CUDAREAL sx = sdet_vectors[pid_x];
           CUDAREAL sy = sdet_vectors[pid_y];
           CUDAREAL sz = sdet_vectors[pid_z];

           CUDAREAL ox = odet_vectors[pid_x];
           CUDAREAL oy = odet_vectors[pid_y];
           CUDAREAL oz = odet_vectors[pid_z];

           CUDAREAL p0x =pix0_vectors[pid_x];
           CUDAREAL p0y =pix0_vectors[pid_y];
           CUDAREAL p0z =pix0_vectors[pid_z];

           VEC3 _o_vec(ox, oy, oz);

    for(int _thick_tic=0;_thick_tic<detector_thicksteps;++_thick_tic){
           CUDAREAL _Odet = _thick_tic*detector_thickstep;

           CUDAREAL pixposX = _Fdet*fx + _Sdet*sx + _Odet*ox + p0x;
           CUDAREAL pixposY = _Fdet*fy + _Sdet*sy + _Odet*oy + p0y;
           CUDAREAL pixposZ = _Fdet*fz + _Sdet*sz + _Odet*oz + p0z;
           VEC3 _pixel_pos(pixposX, pixposY, pixposZ);

           CUDAREAL _airpath = _pixel_pos.norm();
           VEC3 _diffracted = _pixel_pos/_airpath;

           // solid angle subtended by a pixel: (pix/airpath)^2*cos(2theta)
           CUDAREAL _omega_pixel = pixel_size*pixel_size/_airpath/_airpath*close_distance/_airpath;

           // option to turn off obliquity effect, inverse-square-law only
           if(_point_pixel) _omega_pixel = 1.0/_airpath/_airpath;

           // now calculate detector thickness effects
           CUDAREAL _capture_fraction = 1;

           if(_detector_thick > 0.0 && detector_attnlen > 0.0)
           {
               // inverse of effective thickness increase
               CUDAREAL _parallax = _diffracted.dot(_o_vec) ; //dot_product(diffracted,odet_vector);
               _capture_fraction = exp(-_thick_tic*detector_thickstep/detector_attnlen/_parallax)
                                 -exp(-(_thick_tic+1)*detector_thickstep/detector_attnlen/_parallax);
           }

           // TODO source loop

      for(int _source=0;_source<sources;++_source){
           //VEC3 _incident(-__ldg(&source_X[_source]),
           //               -__ldg(&source_Y[_source]),
           //               -__ldg(&source_Z[_source]));
           VEC3 _incident(-source_X[_source],
                          -source_Y[_source],
                          -source_Z[_source]);
           CUDAREAL _lambda =source_lambda[_source];
           //CUDAREAL _lambda = __ldg(&source_lambda[_source]);
           CUDAREAL lambda_ang = _lambda*1e10;
           if (use_lambda_coefficients){
               lambda_ang = lambda0 + lambda1*lambda_ang;
               _lambda = lambda_ang*1e-10;
           }

           CUDAREAL _source_path = _incident.norm();
           _incident /= _source_path;

           VEC3 _scattering = (_diffracted - _incident) / _lambda;

           CUDAREAL _stol = 0.5*(_scattering.norm()); //magnitude(scattering);

           VEC3 q_vec(_scattering[0], _scattering[1], _scattering[2]);
           q_vec *= 1e-10;

     for (int _sausage_tic=0; _sausage_tic< num_sausages; ++_sausage_tic){

          MAT3 U = sausages_U[_sausage_tic];

    for(int _mos_tic=0;_mos_tic<mosaic_domains;++_mos_tic){
          int amat_idx = mosaic_domains*_sausage_tic+_mos_tic;
          MAT3 UBO = Amatrices[amat_idx];

          VEC3 H_vec = UBO*q_vec;
          CUDAREAL _h = H_vec[0];
          CUDAREAL _k = H_vec[1];
          CUDAREAL _l = H_vec[2];

          int _h0 = ceil(_h - 0.5);
          int _k0 = ceil(_k - 0.5);
          int _l0 = ceil(_l - 0.5);

          VEC3 H0(_h0, _k0, _l0);

          VEC3 delta_H = H_vec - H0;
          VEC3 V = _NABC*delta_H;
          CUDAREAL _hrad_sqr = V.dot(V);
          CUDAREAL exparg = _hrad_sqr/0.63*fudge;
          CUDAREAL _F_latt =0;
          if (exparg< 35) // speed things up?
              _F_latt = Na*Nb*Nc*exp(-exparg);

          //if(_F_latt == 0.0 && ! only_save_omega_kahn) {
          //    continue;
          //}
          CUDAREAL _F_cell = _default_F;
          CUDAREAL _F_cell2 = 0;

          if ( (_h0<=h_max) && (_h0>=h_min) && (_k0<=k_max) && (_k0>=k_min) && (_l0<=l_max) && (_l0>=l_min)  ) {
              int Fhkl_linear_index = (_h0-h_min) * k_range * l_range + (_k0-k_min) * l_range + (_l0-l_min);
              //_F_cell = __ldg(&_FhklLinear[Fhkl_linear_index]);
              _F_cell = _FhklLinear[Fhkl_linear_index];
              //if (complex_miller) _F_cell2 = __ldg(&_Fhkl2Linear[Fhkl_linear_index]);
              if (complex_miller) _F_cell2 = _Fhkl2Linear[Fhkl_linear_index];
          }

          if (complex_miller)
            _F_cell = sqrt(_F_cell*_F_cell + _F_cell2*_F_cell2);

          if (!_oversample_omega)
              _omega_pixel = 1;

          //CUDAREAL sI = __ldg(&source_I[_source]);
          //CUDAREAL Iincrement = _F_cell*_F_cell*_F_latt*_F_latt*sI*_capture_fraction*_omega_pixel;
          CUDAREAL Iincrement = _F_cell*_F_cell*_F_latt*_F_latt*source_I[_source]*_capture_fraction*_omega_pixel;
          //CUDAREAL texture_scale= __ldg(&sausages_scale[_sausage_tic]);
          CUDAREAL texture_scale= sausages_scale[_sausage_tic];
          Iincrement *= texture_scale*texture_scale;
          _I += Iincrement;

          if(verbose > 3)
              printf("hkl= %f %f %f  hkl1= %d %d %d  Fcell=%f\n", _h,_k,_l,_h0,_k0,_l0, _F_cell);

          MAT3 UBOt; //  = U*Bmat_realspace*(eig_O.transpose());
          if (refine_Umat[0]){
              MAT3 RyRzUBOt = RotMats[1]*RotMats[2]*UBOt;
              VEC3 delta_H_prime = (UMATS[_mos_tic]*dRotMats[0]*RyRzUBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;
              CUDAREAL value2 =0;
              if (compute_curvatures) {
                  VEC3 delta_H_dbl_prime = (UMATS[_mos_tic]*d2RotMats[0]*RyRzUBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[0] += value;
              rot_manager_dI2[0] += value2;
          }
          if (refine_Umat[1]){
              MAT3 UmosRx = UMATS[_mos_tic]*RotMats[0];
              MAT3 RzUBOt = RotMats[2]*UBOt;
              VEC3 delta_H_prime =(UmosRx*dRotMats[1]*RzUBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;

              CUDAREAL value2=0;
              if (compute_curvatures){
                  VEC3 delta_H_dbl_prime = (UmosRx*d2RotMats[1]*RzUBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[1] += value;
              rot_manager_dI2[1] += value2;
          }
          if (refine_Umat[2]){
              MAT3 UmosRxRy = UMATS[_mos_tic]*RotMats[0]*RotMats[1];
              VEC3 delta_H_prime = (UmosRxRy*dRotMats[2]*UBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;

              CUDAREAL value2=0;
              if (compute_curvatures){
                  VEC3 delta_H_dbl_prime = (UmosRxRy*d2RotMats[2]*UBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[2] += value;
              rot_manager_dI2[2] += value2;
          }
          //Checkpoint for unit cell derivatives
          //MAT3 Ot = eig_O.transpose();
          for(int i_uc=0; i_uc < 6; i_uc++ ){
              if (refine_Bmat[i_uc]){
                  MAT3 UmosRxRyRzU = UMATS_RXYZ[_mos_tic]*U;
                  VEC3 delta_H_prime = ((UmosRxRyRzU*(dB_mats[i_uc])*s_Ot).transpose()*q_vec);
                  CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                  CUDAREAL value = -two_C * V_dot_dV * Iincrement;
                  CUDAREAL value2 =0;
                  if (compute_curvatures){
                      VEC3 delta_H_dbl_prime = ((UmosRxRyRzU*(dB2_mats[i_uc])*s_Ot).transpose()*q_vec);
                      CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                      CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                      value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                  }
                  ucell_manager_dI[i_uc] += value;
                  ucell_manager_dI2[i_uc] += value2;
              }
          } //end ucell deriv

          // Checkpoint for Ncells manager
          if (refine_Ncells[0]){
              int num_ncell_deriv = 1;
              if (not isotropic_ncells)
                  num_ncell_deriv = 3;
              for (int i_nc=0; i_nc < num_ncell_deriv; i_nc++) {
                  MAT3 dN;
                  dN << 0,0,0,0,0,0,0,0,0;
                  dN(i_nc, i_nc) = 1;
                  CUDAREAL N_i = _NABC(i_nc, i_nc);
                  VEC3 dV_dN = dN*delta_H;
                  CUDAREAL deriv_coef = 1/N_i - C* ( dV_dN.dot(V));
                  CUDAREAL value = 2*Iincrement*deriv_coef;
                  CUDAREAL value2=0;
                  if(compute_curvatures){
                      dN(i_nc, i_nc) = 0; // TODO check maths
                      value2 = ( -1/N_i/N_i - C*(dV_dN.dot(dV_dN))) *2*Iincrement;
                      value2 += deriv_coef*2*value;
                  }
                  Ncells_manager_dI[i_nc] += value;
                  Ncells_manager_dI2[i_nc] += value2;
              }

          } // end Ncells manager deriv

          // Checkpoint for Origin manager
          for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
              if (refine_panel_origin[i_pan_orig]){
                  CUDAREAL per_k = 1/_airpath;
                  CUDAREAL per_k3 = pow(per_k,3);
                  CUDAREAL per_k5 = pow(per_k,5);
                  CUDAREAL lambda_ang = _lambda*1e10;

                  MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                  VEC3 dk;
                  if (i_pan_orig == 0)
                      dk << 0,0,1;
                  else if (i_pan_orig == 1)
                      dk << 1,0,0;
                  else
                      dk << 0,1,0;

                  CUDAREAL G = dk.dot(_pixel_pos);
                  CUDAREAL pix2 = subpixel_size*subpixel_size;
                  VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                  CUDAREAL coef = (M*dk_hat).dot(V);
                  CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                  coef2 += pix2*per_k3*(_o_vec.dot(dk));
                  CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                  pan_orig_manager_dI[i_pan_orig] += value;
                  pan_orig_manager_dI2[i_pan_orig] += 0;

              } // end origin manager deriv
          }

          for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
              if(refine_panel_rot[i_pan_rot]){
                  CUDAREAL per_k = 1/_airpath;
                  CUDAREAL per_k3 = pow(per_k,3);
                  CUDAREAL per_k5 = pow(per_k,5);
                  CUDAREAL lambda_ang = _lambda*1e10;
                  MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                  VEC3 dk = _Fdet*(dF_vecs[_pid*3 + i_pan_rot]) + _Sdet*(dS_vecs[_pid*3 + i_pan_rot]);
                  CUDAREAL G = dk.dot(_pixel_pos);
                  CUDAREAL pix2 = subpixel_size*subpixel_size;
                  VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                  CUDAREAL coef = (M*dk_hat).dot(V);
                  CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                  coef2 += pix2*per_k3*(_o_vec.dot(dk));
                  CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                  pan_rot_manager_dI[i_pan_rot] += value;
                  pan_rot_manager_dI2[i_pan_rot] += 0;
              }
          }

          // checkpoint for Fcell manager
          if (refine_fcell){
              CUDAREAL value = 2*Iincrement/_F_cell ;
              CUDAREAL value2=0;
              if (compute_curvatures){
                  value2 = value/_F_cell;
              }
              fcell_manager_dI += value;
              fcell_manager_dI2 += value2;
          } // end of fcell man deriv

          // checkpoint for eta manager
          if (refine_eta){
              VEC3 DeltaH_deriv = (UMATS_RXYZ_prime[_mos_tic]*UBOt).transpose()*q_vec;
              // vector V is _Nabc*Delta_H
              CUDAREAL value = -two_C*(V.dot(_NABC*DeltaH_deriv))*Iincrement;
              eta_manager_dI += value;
          } // end of eta man deriv

            // sausage deriv
          if (refine_sausages){
              MAT3 UBOt = eig_U*Bmat_realspace*(eig_O.transpose());
              int x = _sausage_tic*3;
              int y = _sausage_tic*3+1;
              int z = _sausage_tic*3+2;
              double value=0;
              for (int i=0;i<3; i++){
                  MAT3 UprimeBOt;
                  if (i==0)
                      UprimeBOt = d_sausages_RXYZ[x] * sausages_RXYZ[y] * sausages_RXYZ[z] * UBOt;
                  else if (i==1)
                      UprimeBOt = sausages_RXYZ[x] * d_sausages_RXYZ[y] * sausages_RXYZ[z] * UBOt;
                  else
                      UprimeBOt = sausages_RXYZ[x] * sausages_RXYZ[y] * d_sausages_RXYZ[z] * UBOt;

                  VEC3 DeltaH_deriv = (UMATS_RXYZ[_mos_tic]*UprimeBOt).transpose()*q_vec;
                  value = -two_C*(V.dot(_NABC*DeltaH_deriv))*Iincrement;
                  sausage_manager_dI[_sausage_tic*4 + i] += value;
              }
              // sausage scale derivative
              value = 2* Iincrement / sausages_scale[_sausage_tic];
              sausage_manager_dI[_sausage_tic*4 + 3] += value;
          }
          // end of sausage deriv

          // checkpoint for lambda manager
          for(int i_lam=0; i_lam < 2; i_lam++){
              if (refine_lambda[i_lam]){
                  CUDAREAL lambda_ang = _lambda*1e10;
                  CUDAREAL NH_dot_V = (_NABC*H_vec).dot(V);
                  CUDAREAL dg_dlambda;
                  if (i_lam==0)
                      dg_dlambda = 1;
                  else // i_lam==1
                      dg_dlambda = lambda_ang;
                  CUDAREAL coef = NH_dot_V*two_C*(dg_dlambda) / lambda_ang;
                  CUDAREAL value = coef*Iincrement;
                  CUDAREAL value2 = 0;
                  lambda_manager_dI[i_lam] += value;
                  lambda_manager_dI2[i_lam] += value2;
              }
          }
          //end of lambda deriv
          if( _printout){
           if( _subS==0 && _subF==0 && _thick_tic==0 && _source==0 &&  _mos_tic==0 && _sausage_tic==0){
            if((_fpixel==_printout_fpixel && _spixel==_printout_spixel) || _printout_fpixel < 0){
               //if( _i_step==0){
                 printf("%4d %4d : stol = %g, lambda = %g\n", _fpixel,_spixel,_stol, _lambda);
                 printf("at %g %g %g\n", _pixel_pos[0],_pixel_pos[1],_pixel_pos[2]);
                 printf("Fdet= %g; Sdet= %g ; Odet= %g\n", _Fdet, _Sdet, _Odet);
                 printf("PIX0: %f %f %f\n" , pix0_vectors[pid_x], pix0_vectors[pid_y], pix0_vectors[pid_z]);
                 printf("F: %f %f %f\n" , fdet_vectors[pid_x], fdet_vectors[pid_y], fdet_vectors[pid_z]);
                 printf("S: %f %f %f\n" , sdet_vectors[pid_x], sdet_vectors[pid_y], sdet_vectors[pid_z]);
                 printf("O: %f %f %f\n" , odet_vectors[pid_x], odet_vectors[pid_y], odet_vectors[pid_z]);
                 printf("pid_x=%d, pid_y=%d; pid_z=%d\n", pid_x, pid_y, pid_z);

                 printf("QVECTOR: %f %f %f\n" , q_vec[0], q_vec[1], q_vec[2]);
                 MAT3 UU = UMATS_RXYZ[_mos_tic];
                   printf("UMAT_RXYZ :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                    UU(0,0),  UU(0,1), UU(0,2),
                    UU(1,0),  UU(1,1), UU(1,2),
                    UU(2,0),  UU(2,1), UU(2,2));
                 UU = Bmat_realspace;
                   printf("Bmat_realspace :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                    UU(0,0),  UU(0,1), UU(0,2),
                    UU(1,0),  UU(1,1), UU(1,2),
                    UU(2,0),  UU(2,1), UU(2,2));
                 UU = UBO;
                   printf("UBO :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                    UU(0,0),  UU(0,1), UU(0,2),
                    UU(1,0),  UU(1,1), UU(1,2),
                    UU(2,0),  UU(2,1), UU(2,2));
                 //printf("source XYZ %g %g %g\n", source_X[0],source_Y[0],source_Z[0]);
                 printf("hkl= %f %f %f  hkl0= %d %d %d\n", _h,_k,_l,_h0,_k0,_l0);
                 printf(" F_cell=%g  F_latt=%g   I = %g\n", _F_cell,_F_latt,_I);
                 printf("I/steps %15.10g\n", _I/Nsteps);
                 printf("omega   %15.10g\n", _omega_pixel);
                 printf("default_F= %f\n", _default_F);
                 printf("Incident[0]=%g, Incident[1]=%g, Incident[2]=%g\n", _incident[0], _incident[1], _incident[2]);
                 printf("source_path %g\n", _source_path);
                 //for (int i_saus=0; i_saus<num_sausages; i_saus++){
                 //  printf("Sausages U (i_sausage=%d, scale=%f) :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                 //   i_saus,sausages_scale[i_saus],
                 //   sausages_U[i_saus](0,0),  sausages_U[i_saus](0,1), sausages_U[i_saus](0,2),
                 //   sausages_U[i_saus](1,0),  sausages_U[i_saus](1,1), sausages_U[i_saus](1,2),
                 //   sausages_U[i_saus](2,0),  sausages_U[i_saus](2,1), sausages_U[i_saus](2,2));
                 //}
              }
            }
          }

       //} // end of i_steps loop
             }
            }
           }
          }
         }
        }
       //} // leaving out olf phi

       CUDAREAL _Fdet_ave = pixel_size*_fpixel + pixel_size/2.0;
       CUDAREAL _Sdet_ave = pixel_size*_spixel + pixel_size/2.0;
       CUDAREAL _Odet_ave = 0; //Odet; // TODO maybe make this more general for thick detectors?

       VEC3 _pixel_pos_ave(0,0,0);
       int pid_x = _pid*3;
       int pid_y = _pid*3+1;
       int pid_z = _pid*3+2;
       //CUDAREAL fx = detector_vectors[pid_x];
       //CUDAREAL fy = detector_vectors[pid_y];
       //CUDAREAL fz = detector_vectors[pid_z];

       //CUDAREAL sx = detector_vectors[stride+pid_x];
       //CUDAREAL sy = detector_vectors[stride+pid_y];
       //CUDAREAL sz = detector_vectors[stride+pid_z];

       //CUDAREAL ox = detector_vectors[stride*2+pid_x];
       //CUDAREAL oy = detector_vectors[stride*2+pid_y];
       //CUDAREAL oz = detector_vectors[stride*2+pid_z];

       //CUDAREAL p0x = detector_vectors[stride*3+pid_x];
       //CUDAREAL p0y = detector_vectors[stride*3+pid_y];
       //CUDAREAL p0z = detector_vectors[stride*3+pid_z];

           CUDAREAL fx = fdet_vectors[pid_x];
           CUDAREAL fy = fdet_vectors[pid_y];
           CUDAREAL fz = fdet_vectors[pid_z];

           CUDAREAL sx = sdet_vectors[pid_x];
           CUDAREAL sy = sdet_vectors[pid_y];
           CUDAREAL sz = sdet_vectors[pid_z];

           CUDAREAL ox = odet_vectors[pid_x];
           CUDAREAL oy = odet_vectors[pid_y];
           CUDAREAL oz = odet_vectors[pid_z];

           CUDAREAL p0x =pix0_vectors[pid_x];
           CUDAREAL p0y =pix0_vectors[pid_y];
           CUDAREAL p0z =pix0_vectors[pid_z];


       //CUDAREAL fx = __ldg(&fdet_vectors[pid_x]);
       //CUDAREAL fy = __ldg(&fdet_vectors[pid_y]);
       //CUDAREAL fz = __ldg(&fdet_vectors[pid_z]);

       //CUDAREAL sx = __ldg(&sdet_vectors[pid_x]);
       //CUDAREAL sy = __ldg(&sdet_vectors[pid_y]);
       //CUDAREAL sz = __ldg(&sdet_vectors[pid_z]);

       //CUDAREAL ox = __ldg(&odet_vectors[pid_x]);
       //CUDAREAL oy = __ldg(&odet_vectors[pid_y]);
       //CUDAREAL oz = __ldg(&odet_vectors[pid_z]);

       //CUDAREAL p0x = __ldg(&pix0_vectors[pid_x]);
       //CUDAREAL p0y = __ldg(&pix0_vectors[pid_y]);
       //CUDAREAL p0z = __ldg(&pix0_vectors[pid_z]);

       _pixel_pos_ave[0] = _Fdet_ave * fx+_Sdet_ave*sx+_Odet_ave*ox+p0x;
       _pixel_pos_ave[1] = _Fdet_ave * fy+_Sdet_ave*sy+_Odet_ave*oy+p0y;
       _pixel_pos_ave[2] = _Fdet_ave * fz+_Sdet_ave*sz+_Odet_ave*oz+p0z;

       CUDAREAL _airpath_ave = _pixel_pos_ave.norm();
       VEC3 _diffracted_ave = _pixel_pos_ave/_airpath_ave;
       CUDAREAL _omega_pixel_ave = pixel_size*pixel_size/_airpath_ave/_airpath_ave*close_distance/_airpath_ave;

       CUDAREAL _polar = 1;
       if (!_nopolar){
           //VEC3 _incident(-__ldg(&source_X[0]), -__ldg(&source_Y[0]), -__ldg(&source_Z[0]));
           VEC3 _incident(-source_X[0], -source_Y[0], -source_Z[0]);
           _incident = _incident / _incident.norm();
           // component of diffracted unit vector along incident beam unit vector
           CUDAREAL cos2theta = _incident.dot(_diffracted_ave);
           CUDAREAL cos2theta_sqr = cos2theta*cos2theta;
           CUDAREAL sin2theta_sqr = 1-cos2theta_sqr;

           CUDAREAL _psi=0;
           if(kahn_factor != 0.0){
               // cross product to get "vertical" axis that is orthogonal to the cannonical "polarization"
               VEC3 B_in = _polarization_axis.cross(_incident);
               // cross product with incident beam to get E-vector direction
               VEC3 E_in = _incident.cross(B_in);
               // get components of diffracted ray projected onto the E-B plane
               CUDAREAL _kEi = _diffracted_ave.dot(E_in);
               CUDAREAL _kBi = _diffracted_ave.dot(B_in);
               // compute the angle of the diffracted ray projected onto the incident E-B plane
               _psi = -atan2(_kBi,_kEi);
           }
           // correction for polarized incident beam
           _polar = 0.5*(1.0 + cos2theta_sqr - kahn_factor*cos(2*_psi)*sin2theta_sqr);
       }

       CUDAREAL _om = 1;
       if (!_oversample_omega)
           _om=_omega_pixel_ave;
       // final scale term to being everything to photon number units
       CUDAREAL _scale_term = _r_e_sqr*_fluence*_spot_scale*_polar*_om / Nsteps*num_sausages;

       floatimage[i_pix] = _scale_term*_I;

       // udpate the rotation derivative images*
       for (int i_rot =0 ; i_rot < 3 ; i_rot++){
           if (refine_Umat[i_rot]){
               CUDAREAL value = _scale_term*rot_manager_dI[i_rot];
               CUDAREAL value2 = _scale_term*rot_manager_dI2[i_rot];
               int idx = i_rot*Npix_to_model + i_pix;
               d_Umat_images[idx] = value;
               //d2_Umat_images[idx] = value2;
           }
       } // end rot deriv image increment

       //update the ucell derivative images
       for (int i_uc=0 ; i_uc < 6 ; i_uc++){
           if (refine_Bmat[i_uc]){
               CUDAREAL value = _scale_term*ucell_manager_dI[i_uc];
               CUDAREAL value2 = _scale_term*ucell_manager_dI2[i_uc];
               int idx= i_uc*Npix_to_model + i_pix;
               d_Bmat_images[idx] = value;
               //d2_Bmat_images[idx] = value2;
           }
       }// end ucell deriv image increment

       //update the Ncells derivative image
       if (refine_Ncells[0]){
           CUDAREAL value = _scale_term*Ncells_manager_dI[0];
           CUDAREAL value2 = _scale_term*Ncells_manager_dI2[0];
           int idx = i_pix;
           d_Ncells_images[idx] = value;
           //d2_Ncells_images[idx] = value2;

           if (! isotropic_ncells){
               value = _scale_term*Ncells_manager_dI[1];
               value2 = _scale_term*Ncells_manager_dI2[1];
               idx = Npix_to_model + i_pix;
               d_Ncells_images[idx] = value;
               //d2_Ncells_images[idx] = value2;

               value = _scale_term*Ncells_manager_dI[2];
               value2 = _scale_term*Ncells_manager_dI2[2];
               idx = Npix_to_model*2 + i_pix;
               d_Ncells_images[idx] = value;
               //d2_Ncells_images[idx] = value2;
           }
       }// end Ncells deriv image increment

       // update Fcell derivative image
       if(refine_fcell){
           CUDAREAL value = _scale_term*fcell_manager_dI;
           CUDAREAL value2 = _scale_term*fcell_manager_dI2;
           d_fcell_images[i_pix] = value;
           //d2_fcell_images[i_pix] = value2;
       }// end Fcell deriv image increment

       // update eta derivative image
       if(refine_eta){
           CUDAREAL value = _scale_term*eta_manager_dI;
           CUDAREAL value2 = 0;
           d_eta_images[i_pix] = value;
       }// end eta deriv image increment

       //update the lambda derivative images
       for (int i_lam=0 ; i_lam < 2 ; i_lam++){
           if (refine_lambda[i_lam]){
               CUDAREAL value = _scale_term*lambda_manager_dI[i_lam];
               CUDAREAL value2 = _scale_term*lambda_manager_dI2[i_lam];
               int idx = i_lam*Npix_to_model + i_pix;
               d_lambda_images[idx] = value;
               //d2_lambda_images[idx] = value2;
           }
       }// end lambda deriv image increment

       // sausage increment
       if (refine_sausages){
           for (int i_sausage=0; i_sausage<num_sausages; i_sausage++){
               for (int i=0; i < 4; i++){
                   int sausage_parameter_i = i_sausage*4+i;
                   double value = _scale_term*sausage_manager_dI[sausage_parameter_i];
                   int idx = sausage_parameter_i*Npix_to_model + i_pix;
                   d_sausage_XYZ_scale_images[idx] = value;
                   }
           }
       }
       // end sausage

       for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
           if(refine_panel_rot[i_pan_rot]){
               CUDAREAL value = _scale_term*pan_rot_manager_dI[i_pan_rot];
               CUDAREAL value2 = _scale_term*pan_rot_manager_dI2[i_pan_rot];
               int idx = i_pan_rot*Npix_to_model + i_pix;
               d_panel_rot_images[idx] = value;
               //d2_panel_rot_images[idx] = value2;
           }
       }// end panel rot deriv image increment

       for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
           if(refine_panel_origin[i_pan_orig]){
               CUDAREAL value = _scale_term*pan_orig_manager_dI[i_pan_orig];
               CUDAREAL value2 = _scale_term*pan_orig_manager_dI2[i_pan_orig];
               int idx = i_pan_orig*Npix_to_model + i_pix;
               d_panel_orig_images[idx] = value;
               //d2_panel_orig_images[idx] = value2;
           }//end panel orig deriv image increment
       }
    } // end i_pix loop
}  // END of GPU kernel

