#include "hip/hip_runtime.h"
#include "diffBraggCUDA.h"

__global__
void gpu_sum_over_steps(
        int Npix_to_model, unsigned int* panels_fasts_slows,
        CUDAREAL* floatimage,
        CUDAREAL* d_Umat_images, CUDAREAL* d2_Umat_images,
        CUDAREAL* d_Bmat_images, CUDAREAL* d2_Bmat_images,
        CUDAREAL* d_Ncells_images, CUDAREAL* d2_Ncells_images,
        CUDAREAL* d_fcell_images, CUDAREAL* d2_fcell_images,
        CUDAREAL* d_eta_images,
        CUDAREAL* d_lambda_images, CUDAREAL* d2_lambda_images,
        CUDAREAL* d_panel_rot_images, CUDAREAL* d2_panel_rot_images,
        CUDAREAL* d_panel_orig_images, CUDAREAL* d2_panel_orig_images,
        CUDAREAL* d_sausage_XYZ_scale_images,
        CUDAREAL* d_sausage_XYZ_scale_images2,
        CUDAREAL* d_sausage_XYZ_scale_images3,
        CUDAREAL* d_sausage_XYZ_scale_images4,
        CUDAREAL* d_sausage_XYZ_scale_images5,
        CUDAREAL* d_sausage_XYZ_scale_images6,
        int* subS_pos, int* subF_pos, int* thick_pos,
        int* source_pos, int* phi_pos, int* mos_pos, int* sausage_pos,
        const int Nsteps, int _printout_fpixel, int _printout_spixel, bool _printout, CUDAREAL _default_F,
        int oversample, bool _oversample_omega, CUDAREAL subpixel_size, CUDAREAL pixel_size,
        CUDAREAL detector_thickstep, CUDAREAL _detector_thick, CUDAREAL close_distance, CUDAREAL detector_attnlen,
        bool use_lambda_coefficients, CUDAREAL lambda0, CUDAREAL lambda1,
        MAT3 eig_U, MAT3 eig_O, MAT3 eig_B, MAT3 RXYZ,
        VEC3* dF_vecs,
        VEC3* dS_vecs,
        MAT3* UMATS_RXYZ,
        MAT3* UMATS_RXYZ_prime,
        MAT3* RotMats,
        MAT3* dRotMats,
        MAT3* d2RotMats,
        MAT3* UMATS,
        MAT3* dB_mats,
        MAT3* dB2_mats,
        MAT3* sausages_RXYZ, MAT3* d_sausages_RXYZ, MAT3* sausages_U, CUDAREAL* sausages_scale,
        CUDAREAL* source_X, CUDAREAL* source_Y, CUDAREAL* source_Z, CUDAREAL* source_lambda, CUDAREAL* source_I,
        CUDAREAL kahn_factor,
        CUDAREAL Na, CUDAREAL Nb, CUDAREAL Nc,
        CUDAREAL phi0, CUDAREAL phistep,
        VEC3 spindle_vec, VEC3 _polarization_axis,
        int h_range, int k_range, int l_range,
        int h_max, int h_min, int k_max, int k_min, int l_max, int l_min, CUDAREAL dmin,
        CUDAREAL fudge, bool complex_miller, int verbose, bool only_save_omega_kahn,
        bool isotropic_ncells, bool compute_curvatures,
        CUDAREAL* _FhklLinear, CUDAREAL* _Fhkl2Linear,
        bool* refine_Bmat, bool* refine_Ncells, bool* refine_panel_origin, bool* refine_panel_rot,
        bool refine_fcell, bool* refine_lambda, bool refine_eta, bool* refine_Umat,
        bool refine_sausages, int num_sausages,
        CUDAREAL* fdet_vectors, CUDAREAL* sdet_vectors,
        CUDAREAL* odet_vectors, CUDAREAL* pix0_vectors,
        bool _nopolar, bool _point_pixel, CUDAREAL _fluence, CUDAREAL _r_e_sqr, CUDAREAL _spot_scale)
{ // BEGIN GPU kernel

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = blockDim.x * gridDim.x;
    for (int i_pix=tid; i_pix < Npix_to_model; i_pix+= thread_stride){
       int _pid = panels_fasts_slows[i_pix*3];
       int _fpixel = panels_fasts_slows[i_pix*3+1];
       int _spixel = panels_fasts_slows[i_pix*3+2];

       // reset photon count for this pixel
       CUDAREAL _I=0;

       // reset derivative photon counts for the various parameters
       CUDAREAL rot_manager_dI[3] = {0,0,0};
       CUDAREAL rot_manager_dI2[3] = {0,0,0};
       CUDAREAL ucell_manager_dI[6]= {0,0,0,0,0,0};
       CUDAREAL ucell_manager_dI2[6]= {0,0,0,0,0,0};
       CUDAREAL Ncells_manager_dI[3]= {0,0,0};
       CUDAREAL Ncells_manager_dI2[3]= {0,0,0};
       CUDAREAL pan_orig_manager_dI[3]= {0,0,0};
       CUDAREAL pan_orig_manager_dI2[3]= {0,0,0};
       CUDAREAL pan_rot_manager_dI[3]= {0,0,0};
       CUDAREAL pan_rot_manager_dI2[3]= {0,0,0};
       CUDAREAL fcell_manager_dI=0;
       CUDAREAL fcell_manager_dI2=0;
       CUDAREAL eta_manager_dI = 0;
       CUDAREAL lambda_manager_dI[2] = {0,0};
       CUDAREAL lambda_manager_dI2[2] = {0,0};

       CUDAREAL sausage_manager_dI[24] = {0,0,0,0,0, // TODO use shared memory determined at runtime to increase max sausages
                                          0,0,0,0,0,
                                          0,0,0,0,0,
                                          0,0,0,0,0,
                                          0,0,0,0}; // maximum of 6 sausages!

       for (int _i_step=0; _i_step < Nsteps; _i_step++){

           int _subS = subS_pos[_i_step];
           int _subF = subF_pos[_i_step];
           int _thick_tic = thick_pos[_i_step];
           int _source = source_pos[_i_step];
           int _phi_tic = phi_pos[_i_step];
           int _mos_tic = mos_pos[_i_step];
           int _sausage_tic = sausage_pos[_i_step];

           // absolute mm position on detector (relative to its origin)
           CUDAREAL _Fdet = subpixel_size*(_fpixel*oversample + _subF ) + subpixel_size/2.0;
           CUDAREAL _Sdet = subpixel_size*(_spixel*oversample + _subS ) + subpixel_size/2.0;

           // assume "distance" is to the front of the detector sensor layer
           CUDAREAL _Odet = _thick_tic*detector_thickstep;
           int pid_x = _pid*3;
           int pid_y = _pid*3+1;
           int pid_z = _pid*3+2;
           VEC3 _o_vec(odet_vectors[pid_x], odet_vectors[pid_y], odet_vectors[pid_z]);

           CUDAREAL pixposX = _Fdet*fdet_vectors[pid_x]+_Sdet*sdet_vectors[pid_x]+_Odet*odet_vectors[pid_x]+pix0_vectors[pid_x];
           CUDAREAL pixposY = _Fdet*fdet_vectors[pid_y]+_Sdet*sdet_vectors[pid_y]+_Odet*odet_vectors[pid_y]+pix0_vectors[pid_y];
           CUDAREAL pixposZ = _Fdet*fdet_vectors[pid_z]+_Sdet*sdet_vectors[pid_z]+_Odet*odet_vectors[pid_z]+pix0_vectors[pid_z];
           VEC3 _pixel_pos(pixposX, pixposY, pixposZ);

           CUDAREAL _airpath = _pixel_pos.norm();
           VEC3 _diffracted = _pixel_pos/_airpath;

           // solid angle subtended by a pixel: (pix/airpath)^2*cos(2theta)
           CUDAREAL _omega_pixel = pixel_size*pixel_size/_airpath/_airpath*close_distance/_airpath;

           // option to turn off obliquity effect, inverse-square-law only
           if(_point_pixel) _omega_pixel = 1.0/_airpath/_airpath;

           // now calculate detector thickness effects
           CUDAREAL _capture_fraction = 1;
           if(_detector_thick > 0.0 && detector_attnlen > 0.0)
           {
               // inverse of effective thickness increase
               CUDAREAL _parallax = _diffracted.dot(_o_vec) ; //dot_product(diffracted,odet_vector);
               _capture_fraction = exp(-_thick_tic*detector_thickstep/detector_attnlen/_parallax)
                                 -exp(-(_thick_tic+1)*detector_thickstep/detector_attnlen/_parallax);
           }
           VEC3 _incident(-source_X[_source], -source_Y[_source], -source_Z[_source]);
           CUDAREAL _lambda = source_lambda[_source];
           CUDAREAL lambda_ang = _lambda*1e10;
           if (use_lambda_coefficients){
               lambda_ang = lambda0 + lambda1*lambda_ang;
               _lambda = lambda_ang*1e-10;
           }

           CUDAREAL _source_path = _incident.norm();
           _incident /= _source_path;

           VEC3 _scattering = (_diffracted - _incident) / _lambda;

           CUDAREAL _stol = 0.5*(_scattering.norm()); //magnitude(scattering);

           //if(dmin > 0.0 && _stol > 0.0)
           //{
           //    if(dmin > 0.5/_stol)
           //    {
           //        continue;
           //    }
           //}

          CUDAREAL _phi = phi0 + phistep*_phi_tic;
          MAT3 Bmat_realspace = eig_B;
          if( _phi != 0.0 )
          {
              CUDAREAL cosphi = cos(_phi);
              CUDAREAL sinphi = sin(_phi);
              VEC3 ap_vec(eig_B(0,0), eig_B(1,0), eig_B(2,0));
              VEC3 bp_vec(eig_B(0,1), eig_B(1,1), eig_B(2,1));
              VEC3 cp_vec(eig_B(0,2), eig_B(1,2), eig_B(2,2));

              ap_vec = ap_vec*cosphi + spindle_vec.cross(ap_vec)*sinphi + spindle_vec*(spindle_vec.dot(ap_vec))*(1-cosphi);
              bp_vec = bp_vec*cosphi + spindle_vec.cross(bp_vec)*sinphi + spindle_vec*(spindle_vec.dot(bp_vec))*(1-cosphi);
              cp_vec = cp_vec*cosphi + spindle_vec.cross(cp_vec)*sinphi + spindle_vec*(spindle_vec.dot(cp_vec))*(1-cosphi);

              Bmat_realspace << ap_vec[0], bp_vec[0], cp_vec[0],
                                  ap_vec[1], bp_vec[1], cp_vec[1],
                                  ap_vec[2], bp_vec[2], cp_vec[2];
          }
          Bmat_realspace *= 1e10;

          MAT3 U = sausages_U[_sausage_tic] * eig_U;
          MAT3 UBO = (UMATS_RXYZ[_mos_tic] * U*Bmat_realspace*(eig_O.transpose())).transpose();

          VEC3 q_vec(_scattering[0], _scattering[1], _scattering[2]);
          q_vec *= 1e-10;
          VEC3 H_vec = UBO*q_vec;

          CUDAREAL _h = H_vec[0];
          CUDAREAL _k = H_vec[1];
          CUDAREAL _l = H_vec[2];

          int _h0 = ceil(_h - 0.5);
          int _k0 = ceil(_k - 0.5);
          int _l0 = ceil(_l - 0.5);

          VEC3 H0(_h0, _k0, _l0);
          MAT3 _NABC;
          _NABC << Na,0,0,
                  0,Nb,0,
                  0,0,Nc;

          CUDAREAL C = 2 / 0.63 * fudge;
          VEC3 delta_H = H_vec - H0;
          VEC3 V = _NABC*delta_H;
          CUDAREAL _hrad_sqr = V.dot(V);
          CUDAREAL _F_latt = Na*Nb*Nc*exp(-( _hrad_sqr / 0.63 * fudge ));

          //if(_F_latt == 0.0 && ! only_save_omega_kahn) {
          //    continue;
          //}

          CUDAREAL _F_cell = _default_F;
          CUDAREAL _F_cell2 = 0;

          if ( (_h0<=h_max) && (_h0>=h_min) && (_k0<=k_max) && (_k0>=k_min) && (_l0<=l_max) && (_l0>=l_min)  ) {
              int Fhkl_linear_index = (_h0-h_min) * k_range * l_range + (_k0-k_min) * l_range + (_l0-l_min);
              _F_cell = _FhklLinear[Fhkl_linear_index];
              if (complex_miller) _F_cell2 = _Fhkl2Linear[Fhkl_linear_index];
          }

          if (complex_miller)
            _F_cell = sqrt(_F_cell*_F_cell + _F_cell2*_F_cell2);

          if (!_oversample_omega)
              _omega_pixel = 1;

          CUDAREAL Iincrement = _F_cell*_F_cell*_F_latt*_F_latt*source_I[_source]*_capture_fraction*_omega_pixel;
          Iincrement *= sausages_scale[_sausage_tic]*sausages_scale[_sausage_tic];
          _I += Iincrement;

          if(verbose > 3)
              printf("hkl= %f %f %f  hkl1= %d %d %d  Fcell=%f\n", _h,_k,_l,_h0,_k0,_l0, _F_cell);

          CUDAREAL two_C = 2*C;
          MAT3 UBOt = U*Bmat_realspace*(eig_O.transpose());
          if (refine_Umat[0]){
              MAT3 RyRzUBOt = RotMats[1]*RotMats[2]*UBOt;
              VEC3 delta_H_prime = (UMATS[_mos_tic]*dRotMats[0]*RyRzUBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;
              CUDAREAL value2 =0;
              if (compute_curvatures) {
                  VEC3 delta_H_dbl_prime = (UMATS[_mos_tic]*d2RotMats[0]*RyRzUBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[0] += value;
              rot_manager_dI2[0] += value2;
          }
          if (refine_Umat[1]){
              MAT3 UmosRx = UMATS[_mos_tic]*RotMats[0];
              MAT3 RzUBOt = RotMats[2]*UBOt;
              VEC3 delta_H_prime =(UmosRx*dRotMats[1]*RzUBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;

              CUDAREAL value2=0;
              if (compute_curvatures){
                  VEC3 delta_H_dbl_prime = (UmosRx*d2RotMats[1]*RzUBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[1] += value;
              rot_manager_dI2[1] += value2;
          }
          if (refine_Umat[2]){
              MAT3 UmosRxRy = UMATS[_mos_tic]*RotMats[0]*RotMats[1];
              VEC3 delta_H_prime = (UmosRxRy*dRotMats[2]*UBOt).transpose()*q_vec;
              CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
              CUDAREAL value = -two_C * V_dot_dV * Iincrement;

              CUDAREAL value2=0;
              if (compute_curvatures){
                  VEC3 delta_H_dbl_prime = (UmosRxRy*d2RotMats[2]*UBOt).transpose()*q_vec;
                  CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                  CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                  value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
              }
              rot_manager_dI[2] += value;
              rot_manager_dI2[2] += value2;
          }
          //Checkpoint for unit cell derivatives
          MAT3 Ot = eig_O.transpose();
          for(int i_uc=0; i_uc < 6; i_uc++ ){
              if (refine_Bmat[i_uc]){
                  MAT3 UmosRxRyRzU = UMATS_RXYZ[_mos_tic]*U;
                  VEC3 delta_H_prime = ((UmosRxRyRzU*(dB_mats[i_uc])*Ot).transpose()*q_vec);
                  CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                  CUDAREAL value = -two_C * V_dot_dV * Iincrement;
                  CUDAREAL value2 =0;
                  if (compute_curvatures){
                      VEC3 delta_H_dbl_prime = ((UmosRxRyRzU*(dB2_mats[i_uc])*Ot).transpose()*q_vec);
                      CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                      CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                      value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                  }
                  ucell_manager_dI[i_uc] += value;
                  ucell_manager_dI2[i_uc] += value2;
              }
          } //end ucell deriv

          // Checkpoint for Ncells manager
          if (refine_Ncells[0]){
              int num_ncell_deriv = 1;
              if (not isotropic_ncells)
                  num_ncell_deriv = 3;
              for (int i_nc=0; i_nc < num_ncell_deriv; i_nc++) {
                  MAT3 dN;
                  dN << 0,0,0,0,0,0,0,0,0;
                  dN(i_nc, i_nc) = 1;
                  CUDAREAL N_i = _NABC(i_nc, i_nc);
                  VEC3 dV_dN = dN*delta_H;
                  CUDAREAL deriv_coef = 1/N_i - C* ( dV_dN.dot(V));
                  CUDAREAL value = 2*Iincrement*deriv_coef;
                  CUDAREAL value2=0;
                  if(compute_curvatures){
                      dN(i_nc, i_nc) = 0; // TODO check maths
                      value2 = ( -1/N_i/N_i - C*(dV_dN.dot(dV_dN))) *2*Iincrement;
                      value2 += deriv_coef*2*value;
                  }
                  Ncells_manager_dI[i_nc] += value;
                  Ncells_manager_dI2[i_nc] += value2;
              }

          } // end Ncells manager deriv

          // Checkpoint for Origin manager
          for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
              if (refine_panel_origin[i_pan_orig]){
                  CUDAREAL per_k = 1/_airpath;
                  CUDAREAL per_k3 = pow(per_k,3);
                  CUDAREAL per_k5 = pow(per_k,5);
                  CUDAREAL lambda_ang = _lambda*1e10;

                  MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                  VEC3 dk;
                  if (i_pan_orig == 0)
                      dk << 0,0,1;
                  else if (i_pan_orig == 1)
                      dk << 1,0,0;
                  else
                      dk << 0,1,0;

                  CUDAREAL G = dk.dot(_pixel_pos);
                  CUDAREAL pix2 = subpixel_size*subpixel_size;
                  VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                  CUDAREAL coef = (M*dk_hat).dot(V);
                  CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                  coef2 += pix2*per_k3*(_o_vec.dot(dk));
                  CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                  pan_orig_manager_dI[i_pan_orig] += value;
                  pan_orig_manager_dI2[i_pan_orig] += 0;

              } // end origin manager deriv
          }

          for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
              if(refine_panel_rot[i_pan_rot]){
                  CUDAREAL per_k = 1/_airpath;
                  CUDAREAL per_k3 = pow(per_k,3);
                  CUDAREAL per_k5 = pow(per_k,5);
                  CUDAREAL lambda_ang = _lambda*1e10;
                  MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                  VEC3 dk = _Fdet*(dF_vecs[_pid*3 + i_pan_rot]) + _Sdet*(dS_vecs[_pid*3 + i_pan_rot]);
                  CUDAREAL G = dk.dot(_pixel_pos);
                  CUDAREAL pix2 = subpixel_size*subpixel_size;
                  VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                  CUDAREAL coef = (M*dk_hat).dot(V);
                  CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                  coef2 += pix2*per_k3*(_o_vec.dot(dk));
                  CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                  pan_rot_manager_dI[i_pan_rot] += value;
                  pan_rot_manager_dI2[i_pan_rot] += 0;
              }
          }

          // checkpoint for Fcell manager
          if (refine_fcell){
              CUDAREAL value = 2*Iincrement/_F_cell ;
              CUDAREAL value2=0;
              if (compute_curvatures){
                  value2 = value/_F_cell;
              }
              fcell_manager_dI += value;
              fcell_manager_dI2 += value2;
          } // end of fcell man deriv

          // checkpoint for eta manager
          if (refine_eta){
              VEC3 DeltaH_deriv = (UMATS_RXYZ_prime[_mos_tic]*UBOt).transpose()*q_vec;
              // vector V is _Nabc*Delta_H
              CUDAREAL value = -two_C*(V.dot(_NABC*DeltaH_deriv))*Iincrement;
              eta_manager_dI += value;
          } // end of eta man deriv

            // sausage deriv
          if (refine_sausages){
              MAT3 UBOt = eig_U*Bmat_realspace*(eig_O.transpose());
              int x = _sausage_tic*3;
              int y = _sausage_tic*3+1;
              int z = _sausage_tic*3+2;
              double value=0;
              for (int i=0;i<3; i++){
                  MAT3 UprimeBOt;
                  if (i==0)
                      UprimeBOt = d_sausages_RXYZ[x] * sausages_RXYZ[y] * sausages_RXYZ[z] * UBOt;
                  else if (i==1)
                      UprimeBOt = sausages_RXYZ[x] * d_sausages_RXYZ[y] * sausages_RXYZ[z] * UBOt;
                  else
                      UprimeBOt = sausages_RXYZ[x] * sausages_RXYZ[y] * d_sausages_RXYZ[z] * UBOt;

                  VEC3 DeltaH_deriv = (UMATS_RXYZ[_mos_tic]*UprimeBOt).transpose()*q_vec;
                  value = -two_C*(V.dot(_NABC*DeltaH_deriv))*Iincrement;
                  sausage_manager_dI[_sausage_tic*4 + i] += value;
              }
              // sausage scale derivative
              value = 2* Iincrement / sausages_scale[_sausage_tic];
              sausage_manager_dI[_sausage_tic*4 + 3] += value;
          }
          // end of sausage deriv

          // checkpoint for lambda manager
          for(int i_lam=0; i_lam < 2; i_lam++){
              if (refine_lambda[i_lam]){
                  CUDAREAL lambda_ang = _lambda*1e10;
                  CUDAREAL NH_dot_V = (_NABC*H_vec).dot(V);
                  CUDAREAL dg_dlambda;
                  if (i_lam==0)
                      dg_dlambda = 1;
                  else // i_lam==1
                      dg_dlambda = lambda_ang;
                  CUDAREAL coef = NH_dot_V*two_C*(dg_dlambda) / lambda_ang;
                  CUDAREAL value = coef*Iincrement;
                  CUDAREAL value2 = 0;
                  lambda_manager_dI[i_lam] += value;
                  lambda_manager_dI2[i_lam] += value2;
              }
          }
          //end of lambda deriv
          if( _printout && _i_step==0 ){
              if((_fpixel==_printout_fpixel && _spixel==_printout_spixel) || _printout_fpixel < 0)
              {
                 printf("%4d %4d : stol = %g, lambda = %g\n", _fpixel,_spixel,_stol, _lambda);
                 printf("at %g %g %g\n", _pixel_pos[0],_pixel_pos[1],_pixel_pos[2]);
                 printf("source XYZ %g %g %g\n", source_X[0],source_Y[0],source_Z[0]);
                 printf("hkl= %f %f %f  hkl0= %d %d %d\n", _h,_k,_l,_h0,_k0,_l0);
                 printf(" F_cell=%g  F_latt=%g   I = %g\n", _F_cell,_F_latt,_I);
                 printf("I/steps %15.10g\n", _I/Nsteps);
                 printf("omega   %15.10g\n", _omega_pixel);
                 printf("default_F= %f\n", _default_F);
                 printf("Incident[0]=%g, Incident[1]=%g, Incident[2]=%g\n", _incident[0], _incident[1], _incident[2]);
                 printf("source_path %g\n", _source_path);
                 for (int i_saus=0; i_saus<num_sausages; i_saus++){
                   printf("Sausages U (i_sausage=%d, scale=%f) :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                    i_saus,sausages_scale[i_saus],
                    sausages_U[i_saus](0,0),  sausages_U[i_saus](0,1), sausages_U[i_saus](0,2),
                    sausages_U[i_saus](1,0),  sausages_U[i_saus](1,1), sausages_U[i_saus](1,2),
                    sausages_U[i_saus](2,0),  sausages_U[i_saus](2,1), sausages_U[i_saus](2,2));
                 }
              }
          }

       } // end of i_steps loop

       CUDAREAL _Fdet_ave = pixel_size*_fpixel + pixel_size/2.0;
       CUDAREAL _Sdet_ave = pixel_size*_spixel + pixel_size/2.0;
       CUDAREAL _Odet_ave = 0; //Odet; // TODO maybe make this more general for thick detectors?

       VEC3 _pixel_pos_ave(0,0,0);
       int pid_x = _pid*3;
       int pid_y = _pid*3+1;
       int pid_z = _pid*3+2;
       _pixel_pos_ave[0] = _Fdet_ave * fdet_vectors[pid_x]+_Sdet_ave*sdet_vectors[pid_x]+_Odet_ave*odet_vectors[pid_x]+pix0_vectors[pid_x];
       _pixel_pos_ave[1] = _Fdet_ave * fdet_vectors[pid_y]+_Sdet_ave*sdet_vectors[pid_y]+_Odet_ave*odet_vectors[pid_y]+pix0_vectors[pid_y];
       _pixel_pos_ave[2] = _Fdet_ave * fdet_vectors[pid_z]+_Sdet_ave*sdet_vectors[pid_z]+_Odet_ave*odet_vectors[pid_z]+pix0_vectors[pid_z];

       CUDAREAL _airpath_ave = _pixel_pos_ave.norm();
       VEC3 _diffracted_ave = _pixel_pos_ave/_airpath_ave;
       CUDAREAL _omega_pixel_ave = pixel_size*pixel_size/_airpath_ave/_airpath_ave*close_distance/_airpath_ave;

       CUDAREAL _polar = 1;
       if (!_nopolar){
           VEC3 _incident(-source_X[0], -source_Y[0], -source_Z[0]);
           _incident = _incident / _incident.norm();
           // component of diffracted unit vector along incident beam unit vector
           CUDAREAL cos2theta = _incident.dot(_diffracted_ave);
           CUDAREAL cos2theta_sqr = cos2theta*cos2theta;
           CUDAREAL sin2theta_sqr = 1-cos2theta_sqr;

           CUDAREAL _psi=0;
           if(kahn_factor != 0.0){
               // cross product to get "vertical" axis that is orthogonal to the cannonical "polarization"
               VEC3 B_in = _polarization_axis.cross(_incident);
               // cross product with incident beam to get E-vector direction
               VEC3 E_in = _incident.cross(B_in);
               // get components of diffracted ray projected onto the E-B plane
               CUDAREAL _kEi = _diffracted_ave.dot(E_in);
               CUDAREAL _kBi = _diffracted_ave.dot(B_in);
               // compute the angle of the diffracted ray projected onto the incident E-B plane
               _psi = -atan2(_kBi,_kEi);
           }
           // correction for polarized incident beam
           _polar = 0.5*(1.0 + cos2theta_sqr - kahn_factor*cos(2*_psi)*sin2theta_sqr);
       }

       CUDAREAL _om = 1;
       if (!_oversample_omega)
           _om=_omega_pixel_ave;
       // final scale term to being everything to photon number units
       CUDAREAL _scale_term = _r_e_sqr*_fluence*_spot_scale*_polar*_om / Nsteps*num_sausages;

       floatimage[i_pix] = _scale_term*_I;

       // udpate the rotation derivative images*
       for (int i_rot =0 ; i_rot < 3 ; i_rot++){
           if (refine_Umat[i_rot]){
               CUDAREAL value = _scale_term*rot_manager_dI[i_rot];
               CUDAREAL value2 = _scale_term*rot_manager_dI2[i_rot];
               int idx = i_rot*Npix_to_model + i_pix;
               d_Umat_images[idx] = value;
               //d2_Umat_images[idx] = value2;
           }
       } // end rot deriv image increment

       //update the ucell derivative images
       for (int i_uc=0 ; i_uc < 6 ; i_uc++){
           if (refine_Bmat[i_uc]){
               CUDAREAL value = _scale_term*ucell_manager_dI[i_uc];
               CUDAREAL value2 = _scale_term*ucell_manager_dI2[i_uc];
               int idx= i_uc*Npix_to_model + i_pix;
               d_Bmat_images[idx] = value;
               //d2_Bmat_images[idx] = value2;
           }
       }// end ucell deriv image increment

       //update the Ncells derivative image
       if (refine_Ncells[0]){
           CUDAREAL value = _scale_term*Ncells_manager_dI[0];
           CUDAREAL value2 = _scale_term*Ncells_manager_dI2[0];
           int idx = i_pix;
           d_Ncells_images[idx] = value;
           //d2_Ncells_images[idx] = value2;

           if (! isotropic_ncells){
               value = _scale_term*Ncells_manager_dI[1];
               value2 = _scale_term*Ncells_manager_dI2[1];
               idx = Npix_to_model + i_pix;
               d_Ncells_images[idx] = value;
               //d2_Ncells_images[idx] = value2;

               value = _scale_term*Ncells_manager_dI[2];
               value2 = _scale_term*Ncells_manager_dI2[2];
               idx = Npix_to_model*2 + i_pix;
               d_Ncells_images[idx] = value;
               //d2_Ncells_images[idx] = value2;
           }
       }// end Ncells deriv image increment

       // update Fcell derivative image
       if(refine_fcell){
           CUDAREAL value = _scale_term*fcell_manager_dI;
           CUDAREAL value2 = _scale_term*fcell_manager_dI2;
           d_fcell_images[i_pix] = value;
           //d2_fcell_images[i_pix] = value2;
       }// end Fcell deriv image increment

       // update eta derivative image
       if(refine_eta){
           CUDAREAL value = _scale_term*eta_manager_dI;
           CUDAREAL value2 = 0;
           d_eta_images[i_pix] = value;
       }// end eta deriv image increment

       //update the lambda derivative images
       for (int i_lam=0 ; i_lam < 2 ; i_lam++){
           if (refine_lambda[i_lam]){
               CUDAREAL value = _scale_term*lambda_manager_dI[i_lam];
               CUDAREAL value2 = _scale_term*lambda_manager_dI2[i_lam];
               int idx = i_lam*Npix_to_model + i_pix;
               d_lambda_images[idx] = value;
               //d2_lambda_images[idx] = value2;
           }
       }// end lambda deriv image increment

       // sausage increment
       if (refine_sausages){
           for (int i_sausage=0; i_sausage<num_sausages; i_sausage++){
               for (int i=0; i < 4; i++){
                   int sausage_parameter_i = i_sausage*4+i;
                   double value = _scale_term*sausage_manager_dI[sausage_parameter_i];
                   //int idx = sausage_parameter_i*Npix_to_model + i_pix;
                   int idx = i*Npix_to_model + i_pix;
                   if (i_sausage==0)
                       d_sausage_XYZ_scale_images[idx] = value;
                   else if (i_sausage==1)
                       d_sausage_XYZ_scale_images2[idx] = value;
                   else if (i_sausage==2)
                       d_sausage_XYZ_scale_images3[idx] = value;
                   else if (i_sausage==3)
                       d_sausage_XYZ_scale_images4[idx] = value;
                   else if (i_sausage==4)
                       d_sausage_XYZ_scale_images5[idx] = value;
                   else if (i_sausage==5)
                       d_sausage_XYZ_scale_images6[idx] = value;
                   }
           }
       }
       // end sausage

       for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
           if(refine_panel_rot[i_pan_rot]){
               CUDAREAL value = _scale_term*pan_rot_manager_dI[i_pan_rot];
               CUDAREAL value2 = _scale_term*pan_rot_manager_dI2[i_pan_rot];
               int idx = i_pan_rot*Npix_to_model + i_pix;
               d_panel_rot_images[idx] = value;
               //d2_panel_rot_images[idx] = value2;
           }
       }// end panel rot deriv image increment

       for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
           if(refine_panel_origin[i_pan_orig]){
               CUDAREAL value = _scale_term*pan_orig_manager_dI[i_pan_orig];
               CUDAREAL value2 = _scale_term*pan_orig_manager_dI2[i_pan_orig];
               int idx = i_pan_orig*Npix_to_model + i_pix;
               d_panel_orig_images[idx] = value;
               //d2_panel_orig_images[idx] = value2;
           }//end panel orig deriv image increment
       }
    } // end i_pix loop
}  // END of GPU kernel

